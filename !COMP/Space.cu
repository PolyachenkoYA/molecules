#include "hip/hip_runtime.h"

/**
 * PolyachYA Corporation.  All rights reserved.
 *
 * Please refer to the PolyachYA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include "Space.cuh"

//-----------------------------------------------------------------
// -------------------------- Space--------------------------------
//-----------------------------------------------------------------
TSpace::TSpace()
{
	srand(time(0));

	n_cr = glob_time = pressure = n = dissipK = TmpStabEps = r_cut2 = localT = dumpDT = endT = dt = eff = R = Tmp = L.x = L.y = L.z = E.x = E.y = 0;
	TmpStabGap = Nslice = Nfrm = Ntot = Fnum = compMode = binOutF = useCuda = Ek_is_valid = Ep_is_valid = P_is_valid = 0;
	hostXreal = devX = hostX = nullptr;
	devV = hostV = nullptr;
	hostF = nullptr;
	hostOldX = nullptr;
	hostVK = hostRK = nullptr;
	devA = hostA = nullptr;
	devE = hostE = nullptr;
	gCondFnc = nullptr;
	devPressure = hostPressure = nullptr;

	time_t now = time(0);
	sessionID = string(ctime(&now));

    ParticleFileExt = "xyz";
	SGname = "particles." + ParticleFileExt;
    PrmFName = "param.dat";
    HeadFName = "head.txt";
    ConvFName = "convert.dat";
    StpFName = "stop";
    LFile = "L.txt";
    EFile = "E.txt";
    TFile = "Time.txt";
    PFile = "Pressure.txt";
    ConditionFNameBase = "condition";
    FramesFilesFolder = "frames";
    compModeStr = "";

    ParamFHead = {"Ntot", "n", "n_cr", "r_cut", "endT", "dumpDT", "dt", "Tmp",
    		      "dissipK", "TmpStabEps", "TmpStabGap", "compMode", "binOutF"};
    GalaxyFHead = {"x", "y", "z", "Vx", "Vy", "Vz", "N"};
}

TSpace::~TSpace(){
	free(hostPressure);
	free(hostX);
	free(hostXreal);
	free(hostV);
	free(hostF);
	free(hostOldX);
	free(hostA);
	free(hostE);
	if(gCondFnc) free(gCondFnc);

	int i;
	if(hostVK) for(i = 0; i < 4; ++i)
		free(hostVK[i]);
	if(hostRK) for(i = 0; i < 4; ++i)
		free(hostRK[i]);
	free(hostVK);
	free(hostRK);

	if(useCuda){
		CUDA_CHECK(hipFree(devPressure));
		CUDA_CHECK(hipFree(devX));
		CUDA_CHECK(hipFree(devV));
		CUDA_CHECK(hipFree(devA));
		CUDA_CHECK(hipFree(devE));
	}
}

int TSpace::sayLog(string s, bool printOnScreen){
	return SayLog(LogFName, s, printOnScreen);
}

int TSpace::AreClosePartcls(double3 c0, int Ncurr)
{
	int i;
	double3 r;
	double D = 2*R;
	for(i = 0; i < Ncurr; ++i){
		r = shiftR(hostX[i] - c0, R, D);
		if(dot(r,r) < 1) // r(x,x0) < a
			return 1;
	}
	return 0;
}

int TSpace::CreateParticles(void)
{
	sayLog("   CreateParticles{\n");
	int i;

	sayLog("      particles generation...\n");
	if(start_crystal_lattice){ // n > n_cr -> cristal
		int i2, i3;

		int k = int(pow(Ntot/4, 1.0/3) + 0.5); // it's ok because we check if N is a perfect cube before
		double l = pow(4/n, 1.0/3);
		int n_shift = k/2;
		for(i = 0; i < k; ++i) for(i2 = 0; i2 < k; ++i2) for(i3 = 0; i3 < k; ++i3) { // fcc lattice
			hostX[(i3 + (i2 + i * k) * k) * 4] =
					make_double3(i - n_shift, i2 - n_shift, i3 - n_shift) * l;
			hostX[(i3 + (i2 + i * k) * k) * 4 + 1] =
					make_double3(i - n_shift + 0.5, i2 - n_shift + 0.5, i3 - n_shift) * l;
			hostX[(i3 + (i2 + i * k) * k) * 4 + 2] =
					make_double3(i - n_shift + 0.5, i2 - n_shift, i3 - n_shift + 0.5) * l;
			hostX[(i3 + (i2 + i * k) * k) * 4 + 3] =
					make_double3(i - n_shift, i2 - n_shift + 0.5, i3 - n_shift + 0.5) * l;
		}

		/*     cubic lattice
		int k = int(pow(Ntot, 1.0/3) + 0.5); // it's ok because we check if N is a perfect cube before
		double l = pow(n, -1.0/3);
		int n_shift = k/2;
		for(i = 0; i < k; ++i) for(i2 = 0; i2 < k; ++i2) for(i3 = 0; i3 < k; ++i3) {
			hostX[i3 + (i2 + i * k) * k] = make_double3(i - n_shift, i2 - n_shift, i3 - n_shift) * l * (1 + SYS_EPS);
		}*/
	} else { // n < n_cr -> random gas
		for(i = 0; i < Ntot; ++i){
			// too close particles can cause huge x''
			do{
				hostX[i] = make_double3(myRnd(-R, R), myRnd(-R, R), myRnd(-R, R)) * (1 - SYS_EPS);
			}while(AreClosePartcls(hostX[i], i));
		}
	}
	for(i = 0; i < Ntot; ++i)
		hostXreal[i] = hostX[i];

	for(i = 0; i < Ntot; ++i){
		hostV[i] = rndVec(gauss3D(Tmp));
	}

	sayLog("      ... particles generation DONE\n");

	i = stopMacroMovement();
	if(i) return i;
	//i = centerCM();
	//if(i) return i;

	sayLog(string("      Center of mass was stabilized\n")+
		   string("   }\n"));

	return 0;
}

int TSpace::findCM(void)
{
	int i;

	Xcm = make_double3(0,0,0);
	for(i = 0; i < Ntot; ++i){
		Xcm += hostX[i];
	}
	Xcm /= Ntot;

	return 0;
}

int TSpace::centerCM(void)
{
	int i, _k = 0;
	double r_av2 = pow(n, -2.0 / 3);
	do{
		findCM();

		for(i = 0; i < Ntot; ++i){
			hostX[i] = shiftR(hostX[i] - Xcm, R);
		}

		++_k;
		if(_k > 100)
			return CantCenterCM;
	}while(dot(Xcm) > r_av2 * 0.0001);

	return 0;
}

int TSpace::stopMacroMovement(void)
{
	int i;

	double3 Vc = make_double3(0,0,0);
	for(i = 0; i < Ntot; ++i){
		Vc += hostV[i];
	}
	Vc /= Ntot;

	for(i = 0; i < Ntot; ++i){
		hostV[i] -= Vc;
	}

	return 0;
}

int TSpace::CheckPartcls(void)
{
	int i,j;
	double3 r;
	for(i = 0; i < Ntot; ++i) for(j = i+1; j < Ntot; ++j){
		r = hostX[i] - hostX[j];
		if(dot(r) < 1){
			return 1;
		}
	}

	return 0;
}

int TSpace::ResizeStars(int _n)
{
	if(_n != Ntot){

		int szf3 = sizeof(double3)*_n;
		int szf2 = sizeof(double2)*_n;
		int szf1 = sizeof(double)*_n;

		hostPressure = (double*)malloc(szf1);
		if(!hostPressure) return NULLmalloc;

		hostX = (double3*)malloc(szf3);
		if(!hostX) return NULLmalloc;

		hostXreal = (double3*)malloc(szf3);
		if(!hostXreal) return NULLmalloc;

		hostV = (double3*)malloc(szf3);
		if(!hostV) return NULLmalloc;

		hostF = (double3*)malloc(szf3);
		if(!hostF) return NULLmalloc;

		hostOldX = (double3*)malloc(szf3);
		if(!hostOldX) return NULLmalloc;

		hostVK = (double3**)malloc(sizeof(double3*)*4);
		if(!hostVK) return NULLmalloc;
		hostRK = (double3**)malloc(sizeof(double3*)*4);
		if(!hostRK) return NULLmalloc;

		for(int i = 0; i < 4; ++i){
			hostVK[i] = (double3*)malloc(szf3);
			if(!hostVK[i]) return NULLmalloc;
			hostRK[i] = (double3*)malloc(szf3);
			if(!hostRK[i]) return NULLmalloc;
		}

		hostA = (double3*)malloc(szf3);
		if(!hostA) return NULLmalloc;

		hostE = (double2*)malloc(szf2);
		if(!hostE) return NULLmalloc;


		if(useCuda){
			CUDA_CHECK(hipMalloc((void**)&devPressure,szf1));
			CUDA_CHECK(hipMalloc((void**)&devX,szf3));
			CUDA_CHECK(hipMalloc((void**)&devV,szf3));
			CUDA_CHECK(hipMalloc((void**)&devA,szf3));
			CUDA_CHECK(hipMalloc((void**)&devE,szf2));
		}

		Ntot = _n;
	}

	return 0;
}

int TSpace::CreateHeadF(string FileName)
{
	sayLog("   CreateHeadF("+FileName+"){\n");
    ofstream Fout(FileName.c_str());
    if(!Fout) return CantCreateFile;
    Fout << Fnum + 1 << "\n"
         << R << "\n"
         << binOutF << "\n"
         << Ntot << "\n"
         << rt << "\n";
    Fout.close();
    sayLog("   }created\n");
    return 0;
}

int TSpace::SaveParams(string FileName)
{
	sayLog("   SaveParams(" + FileName + "){\n");
    int i,spForV = 14;
    ofstream Fout(FileName.c_str());
    if(!Fout){return CantOpenFile;}
    Fout.precision(PrintPres);

    for(i = 0; i < ParamFHead.size(); ++i){ Fout << setw(spForV) << ParamFHead[i]; }
    Fout << '\n';
    sayLog("      text part was written\n");

    vector<double> outData;
    Fout << setw(spForV) << Ntot;
    outData = {n, n_cr, sqrt(r_cut2), endT, 1/dumpDT, 1/dt, Tmp, dissipK, TmpStabEps};
    for(i = 0; i < outData.size(); ++i) Fout << setw(spForV) << outData[i];
    Fout << setw(spForV) << TmpStabGap;
    Fout << setw(spForV) << (compMode - CompModeID) * (useCuda ? -1 : 1);
    Fout << setw(spForV) << binOutF;
    Fout << '\n';
    Fout.close();

    sayLog(string("      numeric part was written\n")+
    	   string("   }\n"));

    return 0;
}

int TSpace::syncForPrint(void)
{
    double dt_d2 = dt/2;
    int i;

    switch(compMode){
    	case LPcompMode:
    		for(i = 0; i < Ntot; ++i){
    			hostV[i] -= hostA[i] * dt_d2;
    		}
    		//sayLog("         velocities were modified for saving\n");
    		break;
    }

    VX_synced_for_print = 1;
    return 0;
}

int TSpace::restoreForComp(void)
{
    double dt_d2 = dt/2;
    int i;

    switch(compMode){
    	case LPcompMode:
    		for(i = 0; i < Ntot; ++i){
    			hostV[i] += hostA[i] * dt_d2;
    		}
    		//sayLog("         velocities were restored\n");
    		break;
    }

    VX_synced_for_print = 0;
    return 0;
}

int TSpace::SaveParticles(string FileName, bool bin, int mode)
{
	sayLog(string("      SaveStars (") + FileName + string("){\n"));
	int i, spForV = 14;
    ofstream Fout;

    if(bin){/*
        Fout.open(FileName.c_str(),ios::binary | ios::trunc);
        if(!Fout){ return CantCreateFile; }
        int buf = Ntot;
        Fout.write((char*)&(buf),sizeof(int));
        Fout.write((char*)&(Fnum),sizeof(int));
        Fout.write((char*)&(totalT),sizeof(double));
        */
    } else {
        Fout.open(FileName.c_str(),ios::trunc);
        if(!Fout){ return CantCreateFile; }
        Fout.precision(PrintPres);
        Fout << Ntot
        	 << "\nLattice=\"1.0 0.0 0.0 0.0 1.0 0.0 0.0 0.0 1.0\" Properties=pos:R:3:velo:R:3 Time=" << toString(glob_time) << "\n";
    }
    sayLog("         Header was written\n");

    //Fout << scientific;
    for(i = 0; i < Ntot; ++i)
    {
    	if(dot(hostX[i]) == INFINITY) return StarXIsInf;
    	if(dot(hostV[i]) == INFINITY) return StarVIsInf;
    	if((hostX[i].x == NAN) || (hostX[i].y == NAN) || (hostX[i].z == NAN) || (hostX[i].x == -NAN) || (hostX[i].y == -NAN) || (hostX[i].z == -NAN)) return StarXIsNan;
    	if((hostV[i].x == NAN) || (hostV[i].y == NAN) || (hostV[i].z == NAN) || (hostV[i].x == -NAN) || (hostV[i].y == -NAN) || (hostV[i].z == -NAN)) return StarVIsNan;
        if(bin){
            Fout.write((char*)&(hostXreal[i].x),sizeof(double));
            Fout.write((char*)&(hostXreal[i].y),sizeof(double));
            Fout.write((char*)&(hostXreal[i].z),sizeof(double));
            Fout.write((char*)&(hostV[i].x),sizeof(double));
            Fout.write((char*)&(hostV[i].y),sizeof(double));
            Fout.write((char*)&(hostV[i].z),sizeof(double));
        } else {
        	Fout << setw(spForV) << hostXreal[i].x
        		 << setw(spForV) << hostXreal[i].y
        		 << setw(spForV) << hostXreal[i].z
        		 << setw(spForV) << hostV[i].x
        		 << setw(spForV) << hostV[i].y
        		 << setw(spForV) << hostV[i].z
        		 << "\n";
        }
    }
    Fout.close();
    sayLog("         Stars were saved(" + FileName + ")\n");
    return 0;
}

int TSpace::checkInput(int _n, double _dumpDT, double _dt)
{
    if(compMode < CompModeID){ compMode += CompModeID; }

    if((compMode != VRcompMode) &&
       (compMode != LPcompMode) &&
       (compMode != RKcompMode) &&
       (compMode != AdVRcompMode) &&
       (compMode != MYcompMode))
      { return WrongCompMode; }
    if(((compMode == RKcompMode) || (compMode == AdVRcompMode))){
    	sayLog("\nRK4 and AdVR schemes are not supported\n", 1);
    	return YetUnsupportedInput;
    }
    if(_n == 1){ sayLog("\nN == 1\n", 1); }
    if(_n <= 0){ return NLessOrEq0; }
    if(_n > MaxN){ return NisTooBig; }
    if(useCuda && (_n % BlockW != 0)){ return CUDA_WRONG_NBlockW; }
    if(_dt <= 0){ return dtLessOrEq0; }
    if(_dt == INFINITY){ return dtIsInf; }
    if(_dumpDT <= 0){ return dumpDTLessOrEq0; }
    if(_dumpDT == INFINITY){ return dumpDTIsInf; }
    if(endT <= 0){ return TLessOrEq0; }
    if(endT == INFINITY){ return TIsInf; }
    //if(dissipK < 0){ return dissipKless0; }
    if(std::abs(dissipK) < SYS_EPS){ sayLog("\n|dissipK| < " + toString(SYS_EPS) + "\n", 1); }
    if(r_cut2 < 0){ return RcutLess0; }
    if(Tmp < 0){ return TmpLess0; } else
    if(Tmp < SYS_EPS){ sayLog("\ninitial temperature < " + toString(SYS_EPS) + "\n", 1); }
    if(TmpStabEps <= 0){ return TmpStabEps_LessEq0; }
    if(TmpStabEps < SYS_EPS){ sayLog("\nTmpStabEps < " + toString(SYS_EPS) + "\n", 1); }
    if(TmpStabGap < 0){ return TmpStabGap_Less0; }
    if(TmpStabGap > 1000000){ sayLog("\nTmpStabGap > 1000000\n", 1); }
	if((n > n_triple * (1 + SYS_EPS)) && !start_crystal_lattice){ sayLog("\nn > " + toString(n_triple) + " but crystal structure is't used. May be impossible to build the system\n", 1); }
    if(start_crystal_lattice){
    	sayLog("\n(n,T) = (" + toString(n) + ";" + toString(Tmp) + "); (n, T)_tp = (" + toString(n_cr) + ";" + toString(Tmp_triple) + ") so cristal structure will be used for initila state\n", 1);
    	if(!isIntPow(_n*2, 3)){
    		return NisntCubeForCristal;
    	}
    	//if(n > 1 + SYS_EPS){
    	//	sayLog("\nn >= 1\n", 1);
    	//	return TooDenseSystem;
    	//}
    }

    if(binOutF == 1){
    	sayLog("\nbin files aren't supported for now\n", 1);
    	return 12345;
    }

    sayLog("         Parameters were checked\n");
    return 0;
}

int TSpace::applyInput(int _n, double _dumpDT, double _dt)
{
	sayLog("      Applying input{\n");

	useCuda = compMode < 0;
	compMode = std::abs(compMode);
	start_crystal_lattice = (n > n_cr * (1 + SYS_EPS)) || (Tmp < 0.7);
	int err = checkInput(_n, _dumpDT, _dt);
	if(err) return err;
	err = ResizeStars(_n);
	if(err) return err;

	nCudaB = (Ntot + BlockW - 1) / BlockW;
	dumpDT = 1.0 / _dumpDT;
	dt = 1.0 / _dt;
	r_cut2 = r_cut2 * r_cut2;

/*	if(n > n_cr * (1 + SYS_EPS)){ // n > n_cr -> cristal
		R = 0.5 * pow(N/n, 1.0/3);
	} else {
		R = pow((Ntot/n), 1.0/3) * 0.5;
	}*/
	R = pow(Ntot/n, 1.0/3) * 0.5;
	if(R < sqrt(r_cut2)){ sayLog("\nR < Rcut\n", 1); }

	sayLog("      }\n");
	return 0;
}

int TSpace::SafeLoadParams(string *goodName)
{
    string b_s = "./" + GlxName + "_" + PrmFName;
    bool b_b = access(b_s.c_str(),0);
	if(b_b){
		sayLog(string("   '" + b_s + "' not found\n") +
		       string("   trying to find './"+PrmFName+"'\n"));
		cout << "'" << b_s << "' not found\n"
			 << "trying to find './" << PrmFName << "' ... ";
		b_s = "./"+PrmFName;
	}
	*goodName = b_s;
	int err = LoadParams(b_s);
    if(b_b) cout << (err ? "failed\n" : "success\n");
    if(err) return err;
    return 0;
}

int TSpace::LoadParams(string FileName)
{
    int _N, err;
    double _dumpDT, _dt;
    string buf_s;
    sayLog("   LoadParams (" + FileName + ") {\n");

    ifstream Fin(FileName.c_str());
    if(!Fin){ return CantOpenFile; }
    std::getline(Fin, buf_s);
    Fin >> _N >> n >> n_cr >> r_cut2
    	>> endT >> _dumpDT >> _dt
    	// endT - total time to compute from 0 to endT
    	// totalT - total time already computed
    	// dumpDT - time between saving to file
    	// dt - time step for computation
    	>> Tmp >> dissipK >> TmpStabEps
    	>> TmpStabGap >> compMode >> binOutF;
    Fin.close();

    sayLog("      Parameters were loaded\n");

    err = applyInput(_N, _dumpDT, _dt);
    if(err) return err;

    sayLog("   }\n   " + string(useCuda ? "GPU" : "CPU") + " in use\n");
    return 0;
}

int TSpace::LoadParticles(string FileName, bool bin, int mode)
{
    int i,N,err;
    string buf;
    ifstream Fin;
    sayLog("   LoadParticles (" + FileName + ") {\n");
    if(bin){
        Fin.open(FileName.c_str(),ios::binary);
        if(!Fin){ return CantOpenFile; }
        Fin.read((char*)&(N),sizeof(int));
    } else {
        Fin.open(FileName.c_str());
        if(!Fin){ return CantOpenFile; }
        Fin >> N;
        getline(Fin,buf);
        getline(Fin,buf);
    }
    if(N <= 0) return NLessOrEq0;
    err = ResizeStars(N);
    if(err) return err;
    sayLog("      Head data was loaded\n");
    for(i = 0; i < Ntot; ++i)
    {
        if(bin){
            Fin.read((char*)&(hostXreal[i].x),sizeof(double));
            Fin.read((char*)&(hostXreal[i].y),sizeof(double));
            Fin.read((char*)&(hostXreal[i].z),sizeof(double));
            Fin.read((char*)&(hostV[i].x),sizeof(double));
            Fin.read((char*)&(hostV[i].y),sizeof(double));
            Fin.read((char*)&(hostV[i].z),sizeof(double));
        } else {
        	Fin >> hostXreal[i].x >> hostXreal[i].y >> hostXreal[i].z >> hostV[i].x >> hostV[i].y >> hostV[i].z;
        }
        hostX[i] = shiftRtrue(hostXreal[i], R);
    }
    Fin.close();
    sayLog("      Stars were loaded\n");

    postGetProc(mode);

    sayLog("   }\n");
    return 0;
}

int TSpace::postGetProc(int mode)
{
	int i;

	if(useCuda){
		CUDA_CHECK(hipMemcpy(devX,hostX,Ntot*sizeof(double3),hipMemcpyHostToDevice));
		sayLog("      devX and devM were filled with host data\n");
	}

	if(mode == CompLoadMode){
		i = useCuda ? GPU_findAllA() : CPU_findAllA();
		if(i) return i;
		sayLog("      Acceleration were computed\n");

		double dt2_d2;
		switch(compMode){
		case VRcompMode:
			dt2_d2 = dt*dt*0.5;
			for(i = 0; i < Ntot; ++i) hostOldX[i] = shiftR(hostX[i] - hostV[i]*dt + hostA[i]*dt2_d2, R);
			sayLog("      oldCrd for VRmode were computed\n");
			break;
		case LPcompMode:
			dt2_d2 = dt*0.5;
			for(i = 0; i < Ntot; ++i) hostV[i] += hostA[i]*dt2_d2;
			sayLog("      v(1/2dt) for LPmode were computed\n");
			break;
		}
	}

	return 0;
}

int TSpace::stabilizeTmp(void)
{
	sayLog("   stabTmp{\n");

	int b_i, i;
	long int print_step, k_step = 0;
	double Tmp_av, std_disp, Tmp_av0;
	double t = 0;
    time_t start_time_glob, start_time_2;
    double *Tmp_inst = new double[TmpStabGap];

    b_i = postGetProc(CompLoadMode);
    if(b_i) return b_i;
    Tmp_av = Tmp;
    print_step = Ntot > 10000 ? 1 : 100000000 / (Ntot * Ntot);


    time(&start_time_glob);
    do
    {
    	b_i = doTimeStep(compMode);
    	if(b_i) return b_i;

    	syncForPrint();
    	computeEk();
    	restoreForComp();

    	Tmp_av = (Tmp_av * (TmpStabGap - 1) + Tmp_curr) / TmpStabGap;
    	Tmp_inst[k_step % TmpStabGap] = Tmp_curr;
		if(k_step == TmpStabGap){
			Tmp_av0 = Tmp_av;
			time(&start_time_2);
		}
    	if(k_step >= TmpStabGap){
    		std_disp = 0;
    		for(i = 0; i < TmpStabGap; ++i){
    			std_disp += pow2(Tmp_av - Tmp_inst[i]);
    		}
    		std_disp = sqrt(std_disp / TmpStabGap) / Tmp_av;

    	}

    	++k_step;
    	t += dt;

        if(k_step % print_step == 0)
        {
        	if(k_step < TmpStabGap){
            	time_progress(start_time_glob, time(0), ((double)k_step / TmpStabGap),
            			"Tmp stabilizing: initial time gap\nT_target = " + toString(Tmp) + "; T_curr_av = " + toString(Tmp_av) + "; T_curr = " + toString(Tmp_curr),
            			1);

        	} else {
            	time_progress(start_time_2, time(0),
            			log(epsDlt(Tmp_av0, Tmp) / epsDlt(Tmp_av, Tmp)) / log(epsDlt(Tmp_av0, Tmp) / TmpStabEps),
            			"Tmp stabilizing: waiting for equilibrium\nT_target = " + toString(Tmp) + "; T_curr_av = " + toString(Tmp_av) + "; T_curr = " + toString(Tmp_curr),
            			1);
        	}
        }
    }while(!(almostEq(Tmp_av, Tmp, TmpStabEps) && almostEq(Tmp_curr, Tmp, TmpStabEps) && (std_disp < TmpStabEps) && (k_step > TmpStabGap)));

    //for(i = 0; i < this->Ntot; ++i){
    //	this->hostV[i] *= sqrt(Tmp / Tmp_curr);
    //}

    delete [] Tmp_inst;
    sayLog("Tmp_target = " + toString(Tmp) + "; Tmp_av = " + toString(Tmp_av) + "; Tmp = " + toString(Tmp_curr) + "                        \n", 1);
    sayLog("   }\n");
    Tmp = Tmp_curr;
    return 0;
}

int TSpace::saveDump(ofstream &FoutT, ofstream &FoutE, ofstream &FoutP, string &FramePath)
{
	int b_i;
	syncForPrint();

	sayLog("      Saving " + toString(++Fnum) + " file...\n");
    FoutT << glob_time << endl;

    b_i = SaveParticles(FramePath + toString(Fnum) + string(".") + ParticleFileExt, binOutF, CompLoadMode);
    if(b_i) return b_i;
    sayLog("      ... saved\n");

    b_i = useCuda ? cudaDoEstep() : doEstep();
    if(b_i) return b_i;

    FoutE << E.x << " " << E.y << " " << (E.x + E.y) << endl;
    sayLog("      Energy computed & saved\n");

	b_i = computePressure();
	if(b_i) return b_i;
    FoutP << pressure << endl;
    sayLog("      Pressure computed & saved\n");

    restoreForComp();

    return 0;
}

int TSpace::main_compute(void)
{
	sayLog("   main_compute{\n");
	if(!access(StpFName.c_str(),0)) return StopFileAlreadyExists;

    int b_i = 0;
    time_t rStime;
    double kt;
    bool stpCalc = 0;
    string buf_s, BaseName = "./" + GlxName + "/";
    string FramePath = BaseName + FramesFilesFolder + "/";

    //ofstream FoutT((BaseName+TFile).c_str(),ios::app);
    // This was to make possible to continue computation, but here it's not possible for other reasons.
    // so no need to keep this bothering reature
    ofstream FoutT((BaseName + TFile).c_str());
    if(!FoutT) return CantCreateFile;
    FoutT.precision(PrintPres);

    ofstream FoutP((BaseName + PFile).c_str());
    if(!FoutP) return CantCreateFile;
    FoutP.precision(PrintPres);

    ofstream FoutE((BaseName + EFile).c_str());
    if(!FoutE) return CantCreateFile;
    FoutE.precision(PrintPres);

    --Fnum;
	b_i = saveDump(FoutT, FoutE, FoutP, FramePath);
	if(b_i) return b_i;

    cout << "Galaxy name : '" << GlxName << "'\n";

    kt = real_time_k();

    sayLog(string("      " + compModeStr + " computation mode\n")+
    	   string("      Starting the computation{\n"));
    time(&rStime);
    do
    {
    	b_i = doTimeStep(compMode);
    	if(b_i) return b_i;
    	localT += dt;
    	glob_time += dt;

        if(localT > dumpDT*(1 - SYS_EPS))
        {
        	localT = 0;

        	b_i = saveDump(FoutT, FoutE, FoutP, FramePath);
        	if(b_i) return b_i;

            time_progress(rStime, time(0), glob_time / endT / kt, "computing");
            stpCalc = !((glob_time < endT - dt*(1 - SYS_EPS)) && access(StpFName.c_str(),0));
        }
    }while(!stpCalc);
    rt = time(0) - rStime;
    FoutT.close();
    FoutP.close();
    FoutE.close();

    eff = glob_time/dt * Ntot*Ntot / rt*3600;
    //if(!useCuda) eff/=2;
    buf_s = glob_time < endT - dt * (1 - SYS_EPS) ? "terminated" : "done";
    buf_s = "Computation was " + buf_s;
    cout << buf_s << "\n";
    sayLog(string("      "+buf_s+"\n")+
    	   string("      }\n")+
    	   string("   }\n")+
    	   string("   Efficiency{\n")+
    	   string("      time computed = "+toString(glob_time)+"; dt = "+toString(dt)+"; usedT/dt = "+toString(glob_time / dt)+"\n")+
           string("      N = " + toString(Ntot) + "; real_t [h] = " + toString(1.0/3600*rt) + "\n")+
           string("      efficiency (e = endT/dt*N^2/t_real) = " + toString(eff) + "\n")+
           string("   }\n"));


    return 0;
}

double TSpace::real_time_k(void)
{
    /*
     * kt is for accounting computeEp effect on time
     * */
    double kt = dt/dumpDT;
    switch(compMode){
		case MYcompMode:
			compModeStr = "MY";
			break;
    	case VRcompMode:
    		compModeStr = "VR";
    		break;
    	case LPcompMode:
    		compModeStr = "LP";
    		break;
    	case RKcompMode:
    		kt /= 4;
    		compModeStr = "RK";
    		break;
    	case AdVRcompMode:
    		kt /= 4;
    		compModeStr = "AdVR";
    		break;
    }
    return 1 + kt * 0.35; // TODO find k - it's not 0.788 here
}

int TSpace::shiftAll(void)
{
	double D = 2*R;
    for(int i = 0; i < Ntot; ++i) hostX[i] = shiftR(hostX[i], R, D);
    return 0;
}

int TSpace::doTimeStep(int cmpMode)
{
    switch (cmpMode){
    	case MYcompMode: stepMY(); break;
        case VRcompMode: stepVR(); break;
        case LPcompMode: stepLP(); break;
        case RKcompMode: stepRK(); break;
        case AdVRcompMode: stepAdVR(); break;
        default: return WrongCompMode;
    }

    // shiftAll();

    // this is necessary here - without it there is a slow drift of CM
    stopMacroMovement();

    Ek_is_valid = 0;
    Ep_is_valid = 0;
    P_is_valid = 0;
    VX_synced_for_print = 0;

    return 0;
}

int TSpace::CPU_findAllA(void)
{
    unsigned long i,j;
    double3 bv, m_x_curr;
    double r2, _f;
    double D = 2 * R;

   for(i = 0; i < Ntot; ++i){
	   hostA[i].x = hostA[i].y = hostA[i].z = 0;
	   hostPressure[i] = 0;
   }

    #ifdef _OPENMP
    	#pragma omp parallel for private(bv, r2, j, m_x_curr, _f)
	#else
		#warning "OpenMP unused"
    #endif
    for(i = 0; i < Ntot; ++i){
    	m_x_curr = -hostX[i];
        for(j = 0; j < i; ++j){
        	 bv = shiftR(hostX[j] + m_x_curr, R, D);
        	 r2 = dot(bv, bv);
        	 if(r2 < r_cut2){
        		 hostA[i] += bv * getForce(r2);
        	 }
        }
        for(j = i + 1; j < Ntot; ++j){
        	 bv = shiftR(hostX[j] + m_x_curr, R, D);
        	 r2 = dot(bv, bv);
        	 if(r2 < r_cut2){
        		 _f = getForce(r2);
        		 hostPressure[i] += r2 * _f;
        		 hostA[i] += bv * _f;
        	 }
        }
        hostPressure[i] = -hostPressure[i];
    }

    return 0;
}

int TSpace::useThermostat(void)
{
    if(thermostatOn){
    	double b_d;
    	int i;

    	b_d = sqrt(2 * Tmp * dissipK / dt);
    	// there shouldn't be *2 in the end. For some reason Temperature converges to Tmp/2 without it.
#ifdef _OPENMP
	#pragma omp parallel for
#else
	#warning "OpenMP unused"
#endif

    	for(i = 0; i < Ntot; ++i){
    		hostA[i] += (make_double3(gaussRand(b_d), gaussRand(b_d), gaussRand(b_d)) - hostV[i] * dissipK);
    	}
    }

    return 0;
}

int TSpace::stepMY(void)
{
    int i;
    double D = 2 * R;
    double3 dx;

    findAllA();
#ifdef _OPENMP
	#pragma omp parallel for
#else
	#warning "OpenMP unused"
#endif
    for(i = 0; i < Ntot; ++i){
    	hostV[i] += hostA[i] * dt;
    	dx = hostV[i] * dt;
    	hostX[i] = shiftR(hostX[i] + dx, R, D);
    	hostXreal[i] += dx;
    }

    return 0;
}

int TSpace::stepVR(void)
{
    int i;
    double3 bv;
    double D = 2*R, dt2 = dt*dt;

    findAllA();
#ifdef _OPENMP
	#pragma omp parallel for private(bv)
#else
	#warning "OpenMP unused"
#endif
    for(i = 0; i < Ntot; ++i){
        bv = hostX[i];
        hostX[i] = shiftR(hostX[i]*2 - hostOldX[i] + hostA[i]*dt2, R, D);
        // here we use 3 points - X[i-1], X[i], X[i+1]. Periodic boundaries are not checked to work correctly
        hostV[i] = shiftR(hostX[i] - hostOldX[i], R, D)/(2*dt);
        hostOldX[i] = bv;
    }
    return 0;
}

int TSpace::stepLP(void)
{
    int i;
    double D = 2*R;
    double3 dx;

    // V is 1/2 time-step ahead of X
    for(i = 0; i < Ntot; ++i){
    	dx = hostV[i] * dt;
    	hostX[i] = shiftR(hostX[i] + dx, R, D); // finish previous step by modifying X
    	hostXreal[i] += dx;
    }
    findAllA();                                        // compute new A
    for(i = 0; i < Ntot; ++i) hostV[i] += hostA[i]*dt; // compute new V

    return 0;
}

int TSpace::stepRK(void)
{
    int i;
    double dt_d2 = dt*0.5;

    findAllA();
#ifdef _OPENMP
	#pragma omp parallel for
#else
	#warning "OpenMP unused"
#endif
    for(i=0;i<Ntot;++i){
    	hostOldX[i] = hostX[i];

    	hostVK[0][i] = hostA[i];
    	hostRK[0][i] = hostV[i];
    	hostX[i] = hostOldX[i] + hostRK[0][i] * dt_d2;
    }

    findAllA();
#ifdef _OPENMP
	#pragma omp parallel for
#else
	#warning "OpenMP unused"
#endif
    for(i=0;i<Ntot;++i){
    	hostVK[1][i] = hostA[i];
    	hostRK[1][i] = hostV[i] + hostVK[0][i] * dt_d2;
    	hostX[i] = hostOldX[i] + hostRK[1][i] * dt_d2;
    }

    findAllA();
#ifdef _OPENMP
	#pragma omp parallel for
#else
	#warning "OpenMP unused"
#endif
    for(i=0;i<Ntot;++i){
    	hostVK[2][i] = hostA[i];
    	hostRK[2][i] = hostV[i] + hostVK[1][i] * dt_d2;
    	hostX[i] = hostOldX[i] + hostRK[2][i] * dt_d2;
    }

    findAllA();
#ifdef _OPENMP
	#pragma omp parallel for
#else
	#warning "OpenMP unused"
#endif
	for(i = 0; i < Ntot; ++i){
    	hostVK[3][i] = hostA[i];
    	hostRK[3][i] = hostV[i] + hostVK[2][i] * dt;
		hostA[i] = (hostVK[0][i] + 2 * hostVK[1][i] + 2 * hostVK[2][i] + hostVK[3][i]) / 6;
		hostX[i] = hostOldX[i] + (hostRK[0][i] + 2 * hostRK[1][i] + 2 * hostRK[2][i] + hostRK[3][i]) * (dt/6);
	}

    for(i = 0; i < Ntot; ++i){
    	hostV[i] += hostA[i] * dt;
    }

    return 0;
}

int TSpace::stepAdVR(void)
{
    int i;
    double k1 = 0.1786178958448091;
    double k2 = -0.2123418310626054;
    double k3 = -0.0662645826698185;

    for(i=0;i<Ntot;++i) hostX[i] += hostV[i]*(k1*dt);
    findAllA();
    for(i=0;i<Ntot;++i){
    	hostV[i] += hostA[i]*((0.5-k2)*dt);
    	hostX[i] += hostV[i]*(dt*k3);
    }
    findAllA();
    for(i=0;i<Ntot;++i){
    	hostV[i] += hostA[i]*(k2*dt);
    	hostX[i] += hostV[i]*((1-2*(k1+k3))*dt);
    }
    findAllA();
    for(i=0;i<Ntot;++i){
    	hostV[i] += hostA[i]*(k2*dt);
    	hostX[i] += hostV[i]*(k3*dt);
    }
    findAllA();
    for(i=0;i<Ntot;++i){
    	hostV[i] += hostA[i]*((0.5-k2)*dt);
    	hostX[i] += hostV[i]*(k1*dt);
    }

    return 0;
}

/*
*/
// TSpace
//------------------------------------------------------------------------------

int TSpace::doCondStep(double dr)
{
	int i, j, k;
	double D = 2*R;

	for(i = 0; i < Nslice; ++i){
		gCondFnc[i] = 0;
	}

	for(i = 0; i < Ntot; ++i){
		for(j = 0; j < i; ++j){
			k = floor( length( shiftR(hostX[j] - hostX[i], R, D) )/dr );
			if(k < Nslice)
				++gCondFnc[k];
		}
		// every particle except i0
		for(i = i+1; i < Ntot; ++i){
			k = floor( length( shiftR(hostX[j] - hostX[i], R, D) )/dr );
			if(k < Nslice)
				++gCondFnc[k];
		}
	}

	double kg = 4 * pi * dr*dr * dr * (Ntot/pow3(2*R));
	for(i = 0; i < Nslice; ++i){
		gCondFnc[i] /= (kg * (i+0.5)*(i+0.5));
	}

	return 0;
}

int TSpace::computeEk(void)
{
	E.x = 0;
	for(int i = 0; i < Ntot; ++i){
		E.x += dot(hostV[i]) * 0.5;
	}
	if(std::abs(E.x) == INFINITY) return EkIsInf;
	if((E.x == NAN) || (E.x == -NAN)) return EkIsNan;

	Tmp_curr = 2.0/3 * E.x/Ntot;

	Ek_is_valid = 1;
	return 0;
}

int TSpace::computePressure(void)
// Forces and Ek are expected to be relevant
{
	if(!Ek_is_valid)
		return NoValidTforP;

	int i;

	pressure = 0;
	for(i = 0; i < Ntot; ++i){
		pressure += hostPressure[i];
	}
	pressure /= (3 * pow3(2 * R));
	pressure += n * Tmp_curr;

	P_is_valid = 1;
	return 0;
}

int TSpace::computeEp(void)
{
	int i, j;
	double3 r, m_x_curr;
	double r2;
	double D = 2*R;

	E.y = 0;
	for(i = 0; i < Ntot; ++i){
		hostE[i].y = 0;
	}

// this openmp is checked - it has no influence on accuracy
#ifdef _OPENMP
	#pragma omp parallel for private(r2, j, r, m_x_curr)
#else
	#warning "OpenMP unused"
#endif
	for(i = 0; i < Ntot; ++i){
		m_x_curr = -hostX[i];
		for(j = i+1; j < Ntot; ++j){
			r = shiftR(hostX[j] + m_x_curr, R, D);
			r2 = dot(r,r);
			if(r2 < r_cut2)
				hostE[i].y += getEp(r2);
		}
	}
	for(i = 0; i < Ntot; ++i) E.y += hostE[i].y;
	//E.y -= getEp(r_cut2) * Ntot * (Ntot - 1) / 2; // substract Ecut

	if(std::abs(E.y) == INFINITY) return EpIsInf;
	if((E.y == NAN) || (E.y == -NAN)) return EpIsNan;

	Ep_is_valid = 1;
	return 0;
}

int TSpace::doEstep(void)
{
	if(!VX_synced_for_print)
		return AttemptToPrintOutsyncedData;

	int err_handl = computeEk();
	if(err_handl) return err_handl;
	err_handl = computeEp();
	if(err_handl) return err_handl;

    return 0;
}

int TSpace::cudaDoEstep(void)
{
	if(!VX_synced_for_print)
		return AttemptToPrintOutsyncedData;

	int i;
	CUDA_CHECK(hipMemcpy(devV, hostV, Ntot * sizeof(double3), hipMemcpyHostToDevice));
	kernel_FindE<<< nCudaB,BlockW >>>(devX, devV, devE, R, r_cut2, Ntot);
	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipMemcpy(hostE, devE, nCudaB * sizeof(double2), hipMemcpyDeviceToHost));
	E.x = E.y = 0;
	for(i = 0; i < nCudaB; ++i) E += hostE[i];
	E.y /= 2; // because we summed all pairs 2 twice in CUDA
	Tmp_curr = 2.0/3 * E.x/Ntot;
	Ek_is_valid = 1;
	Ep_is_valid = 1;

	if(E.x == INFINITY) return EkIsInf;
	if(E.x == NAN) return EkIsNan;
	if(std::abs(E.y) == INFINITY) return EpIsInf;
	if((E.y == NAN) || (E.y == -NAN)) return EpIsNan;

    return 0;
}

__global__ void kernel_FindE(double3 *devX, double3 *devV, double2 *devE, double R, double r_cut2, int Ntot)
{
	__shared__ double3 c1[BlockW], c2[BlockW], v1[BlockW];
	__shared__ double2 e1[BlockW];
	int gind = BlockW * blockIdx.x + threadIdx.x;

	if(gind >= Ntot) return;

	int tind, tile, i, tx = threadIdx.x;
	double3 r;
	double r2, D = 2 * R;
	c1[tx] = -devX[gind];
	v1[tx] = devV[gind];
	e1[tx].x = dot(v1[tx]) * 0.5;
	e1[tx].y = 0;

	for(tile = 0; tile * BlockW + tx < Ntot; ++tile){
		tind = tile * BlockW + tx;
		c2[tx] = devX[tind];
		__syncthreads();
		for(i = 0; i < BlockW; ++i){
			if(tile * BlockW + i != gind){
				r = shiftR(c2[i] + c1[tx], R, D);
				r2 = dot(r, r);
				if(r2 < r_cut2){
					e1[tx].y += getEp(r2);
				}
			}
		}
		__syncthreads();
	}

	i = BlockW / 2;
	while(i > 0){
		if(tx < i){
			tind = tx + i;
			e1[tx].x += e1[tind].x;
			e1[tx].y += e1[tind].y;
		}
		__syncthreads();
		i /= 2;
		/*
		*	For correct work Ntot%BlockW must be ==0, because if it's not, then
		*	/=2 won't properly wrap indexes
		*/
	}

	if(tx==0){
		devE[blockIdx.x].x = e1[0].x;
		devE[blockIdx.x].y = e1[0].y;
	}
}
/*
 */

int TSpace::GPU_findAllA(void)
{
	CUDA_CHECK(hipMemcpy(devX, hostX, Ntot*sizeof(double3), hipMemcpyHostToDevice));
	kernel_FindAllA<<< nCudaB,BlockW >>>(devX, devA, devPressure, R, r_cut2, Ntot);
	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipMemcpy(hostA, devA, Ntot*sizeof(double3), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(hostPressure, devPressure, Ntot*sizeof(double1), hipMemcpyDeviceToHost));

	return 0;
}

int TSpace::findAllA(void)
{
	int res = useCuda ? GPU_findAllA() : CPU_findAllA();
	useThermostat();
	return res;
}

__global__ void kernel_FindAllA(double3 *devX, double3 *devA, double *devPressure, double R, double r_cut2, int Ntot)
{
	__shared__ double3 c1[BlockW], c2[BlockW], a1[BlockW];
	__shared__ double p[BlockW];
	int gind = BlockW * blockIdx.x + threadIdx.x; // global index

	if(gind >= Ntot) return;
	/*
	*	if(Ntot%BlockW != 0) then some threads in one block will never reach
	*	__syncthreads() command because of this "if" statement, and it's bad. So Ntot%BlockW must be == 0
	*/

	// set start values
	int tind, c2_tind, tile, i, tx = threadIdx.x;
	double3 r;
	double _f, r2, D = 2 * R;

	// copy "left column" of stars from global - the ones I'll sum TO
	c1[tx] = -devX[gind];
	a1[tx].x = a1[tx].y = a1[tx].z = 0;
	p[tx] = 0;

	for(tile = 0; tile*BlockW + tx < Ntot; ++tile){
		tind = tile*BlockW + tx;
		// copy part of "up row" from global - the 2nd part of current tile - the ones I'll sum WITH
		c2[tx] = devX[tind];
		// m2[tx] = devM[tind];
		__syncthreads();

		for(i = 0; i < BlockW; ++i){
			c2_tind = tile*BlockW + i;
			if(c2_tind != gind){
				r = shiftR(c2[i] + c1[tx], R, D);
				r2 = dot(r, r);
				if(r2 < r_cut2){
					_f = getForce(r2);
					if(c2_tind > gind)
						p[tx] += r2 * _f;
					a1[tx] += r * _f;
				}
			}
		}

		__syncthreads();
	}

	devA[gind] = a1[tx];
	devPressure[gind] = -p[tx];
}
